#define d_sxx(z,x) d_sxx[(x)*(nz)+(z)]
#define d_szz(z,x) d_szz[(x)*(nz)+(z)]
#define d_Lambda(z,x)  d_Lambda[(x)*(nz)+(z)]
#define d_Cp(z,x)  d_Cp[(x)*(nz)+(z)]

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add_source(float *d_szz, float *d_sxx, float amp, int nz, bool isFor, \
	int z_loc, int x_loc, float dt, float *d_Cp) {

	int id = threadIdx.x + blockDim.x*blockIdx.x;
	float scale = pow(1500.0,2);;
	// float scale = pow(d_Cp(z_loc,x_loc),2);
	// float scale = d_Lambda(z_loc, x_loc);
	// float scale = 10890000.0;
	if (isFor) {
		if(id==0) {	
			// printf("amp = %f  ", amp);
			d_szz(z_loc,x_loc) += scale*amp * dt;
			d_sxx(z_loc,x_loc) += scale*amp * dt;
		}
		else{
			return;
		}
	}
	else {
		if(id==0) {	
			// printf("amp = %f  ", amp);
			d_szz(z_loc,x_loc) -= scale*amp * dt;
			d_sxx(z_loc,x_loc) -= scale*amp * dt;
		}
		else{
			return;
		}
	}
}


// __global__ void add_source(float *d_szz, float *d_sxx, int nz, float *d_source, bool isFor, \
// 	int z_loc, int x_loc, float dt, float *d_Cp) {

// 	int iSrc = threadIdx.x + blockDim.x*blockIdx.x;
// 	float scale = pow(d_Cp(z_loc,x_loc),2);

// 	if (isFor) {
// 		if(id==0) {	
// 			// printf("amp = %f  ", amp);
// 			d_szz(z_loc,x_loc) += scale*amp * dt;
// 			d_sxx(z_loc,x_loc) += scale*amp * dt;
// 		}
// 		else{
// 			return;
// 		}
// 	}
// 	else {
// 		if(id==0) {	
// 			// printf("amp = %f  ", amp);
// 			d_szz(z_loc,x_loc) -= scale*amp * dt;
// 			d_sxx(z_loc,x_loc) -= scale*amp * dt;
// 		}
// 		else{
// 			return;
// 		}
// 	}
// }