#define d_sxx(z,x) d_sxx[(x)*(nz)+(z)]
#define d_szz(z,x) d_szz[(x)*(nz)+(z)]
#define d_Cp(z,x)  d_Cp[(x)*(nz)+(z)]

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void minus_source(float *d_szz, float *d_sxx, float amp, int nz, int z_loc, int x_loc, float dt, float *d_Cp) {
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	float scale = pow(d_Cp(z_loc,x_loc),2);
	// float scale = 10890000.0;
	if(id==0) {	
		// printf("amp = %f  ", amp);
		d_szz(z_loc,x_loc) = d_szz(z_loc,x_loc) - scale*amp * dt;
		d_sxx(z_loc,x_loc) = d_sxx(z_loc,x_loc) - scale*amp * dt;
	}
	else{
		return;
	}
}