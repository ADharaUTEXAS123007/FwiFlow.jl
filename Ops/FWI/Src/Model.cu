#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "Model.h"
#include "Parameter.h"
#include "utilities.h"


// model default constructor
Model::Model() {
    nz_ = 1000;
    nx_ = 1000;
    dim3 threads(TX, TY);
    dim3 blocks((nz_ + TX - 1) / TX, (nx_ + TY - 1) / TY);

    h_Cp = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Cs = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Den = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Lambda = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Mu = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_CpGrad = (float *)malloc(nz_ * nx_ * sizeof(float));

    initialArray(h_Cp, nz_ * nx_, 3300.0);
    // initialArray(h_Cs, nz*nx, 3300.0/sqrt(3.0));
    initialArray(h_Cs, nz_ * nx_, 0.0);
    initialArray(h_Den, nz_ * nx_, 1000.0);
    initialArray(h_Lambda, nz_ * nx_, 0.0);
    initialArray(h_Mu, nz_ * nx_, 0.0);
    initialArray(h_CpGrad, nz_ * nx_, 0.0);


    CHECK(hipMalloc((void **)&d_Cp, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Cs, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Den, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Lambda, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_a, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_b, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_CpGrad, nz_ * nx_ * sizeof(float)));
    intialArrayGPU<<<blocks,threads>>>(d_ave_Mu, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_CpGrad, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_a, nz_, nx_, 1.0/1000.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_b, nz_, nx_, 1.0/1000.0);

    CHECK(hipMemcpy(d_Cp, h_Cp, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Cs, h_Cs, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Den, h_Den, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));

    moduliInit<<< blocks,threads>>>(d_Cp, d_Cs, d_Den, d_Lambda, d_Mu, nz_, nx_);
    aveMuInit<<<blocks,threads>>>(d_Mu, d_ave_Mu, nz_, nx_);
    aveBycInit<<<blocks,threads>>>(d_Den, d_ave_Byc_a, d_ave_Byc_b, nz_, nx_);

    CHECK(hipMemcpy(h_Lambda, d_Lambda, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_Mu, d_Mu, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));


}

// model constructor from parameter file
Model::Model(const Parameter &para, const float *Cp_, const float*Cs_, const float *Den_) {

    nz_ = para.nz();
    nx_ = para.nx();

    dim3 threads(32, 16);
    dim3 blocks((nz_ + 32 - 1) / 32, (nx_ + 16 - 1) / 16);

    h_Cp = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Cs = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Den = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Lambda = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Mu = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_CpGrad = (float *)malloc(nz_ * nx_ * sizeof(float));

    // load Vp, Vs, and Den binaries
    if (para.Cp_fname() == para.Cs_fname()) {
        // only load vp
        fileBinLoad(h_Cp, nz_ * nx_, para.Cp_fname());
        initialArray(h_Cs, nz_ * nx_, 0.0);
        initialArray(h_Den, nz_ * nx_, 1000.0);
    } else {
        for(int i=0;i< nz_*nx_;i++){
            h_Cp[i] = Cp_[i];
            h_Cs[i] = Cs_[i];
            h_Den[i] = Den_[i];
        }
        #if 0
        fileBinLoad(h_Cp, nz_ * nx_, para.Cp_fname());
        fileBinLoad(h_Cs, nz_ * nx_, para.Cs_fname());
        fileBinLoad(h_Den, nz_ * nx_, para.Den_fname());
        #endif
    }
    initialArray(h_Lambda, nz_ * nx_, 0.0);
    initialArray(h_Mu, nz_ * nx_, 0.0);
    initialArray(h_CpGrad, nz_ * nx_, 0.0);


    CHECK(hipMalloc((void **)&d_Cp, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Cs, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Den, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Lambda, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_a, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_b, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_CpGrad, nz_ * nx_ * sizeof(float)));
    intialArrayGPU<<<blocks,threads>>>(d_ave_Mu, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_CpGrad, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_a, nz_, nx_, 1.0/1000.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_b, nz_, nx_, 1.0/1000.0);

    CHECK(hipMemcpy(d_Cp, h_Cp, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Cs, h_Cs, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Den, h_Den, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));

    moduliInit<<<blocks,threads>>>(d_Cp, d_Cs, d_Den, d_Lambda, d_Mu, nz_, nx_);
    aveMuInit<<<blocks,threads>>>(d_Mu, d_ave_Mu, nz_, nx_);
    aveBycInit<<<blocks,threads>>>(d_Den, d_ave_Byc_a, d_ave_Byc_b, nz_, nx_);

    CHECK(hipMemcpy(h_Lambda, d_Lambda, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_Mu, d_Mu, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));

}
// model constructor from parameter file
Model::Model(const Parameter &para) {

    nz_ = para.nz();
    nx_ = para.nx();

    dim3 threads(32, 16);
    dim3 blocks((nz_ + 32 - 1) / 32, (nx_ + 16 - 1) / 16);

    h_Cp = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Cs = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Den = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Lambda = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_Mu = (float *)malloc(nz_ * nx_ * sizeof(float));
    h_CpGrad = (float *)malloc(nz_ * nx_ * sizeof(float));

    // load Vp, Vs, and Den binaries
    if (para.Cp_fname() == para.Cs_fname()) {
        // only load vp
        fileBinLoad(h_Cp, nz_ * nx_, para.Cp_fname());
        initialArray(h_Cs, nz_ * nx_, 0.0);
        initialArray(h_Den, nz_ * nx_, 1000.0);
    } else {
        fileBinLoad(h_Cp, nz_ * nx_, para.Cp_fname());
        fileBinLoad(h_Cs, nz_ * nx_, para.Cs_fname());
        fileBinLoad(h_Den, nz_ * nx_, para.Den_fname());
    }
    initialArray(h_Lambda, nz_ * nx_, 0.0);
    initialArray(h_Mu, nz_ * nx_, 0.0);
    initialArray(h_CpGrad, nz_ * nx_, 0.0);


    CHECK(hipMalloc((void **)&d_Cp, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Cs, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Den, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Lambda, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Mu, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_a, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_ave_Byc_b, nz_ * nx_ * sizeof(float)));
    CHECK(hipMalloc((void **)&d_CpGrad, nz_ * nx_ * sizeof(float)));
    intialArrayGPU<<<blocks,threads>>>(d_ave_Mu, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_CpGrad, nz_, nx_, 0.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_a, nz_, nx_, 1.0/1000.0);
    intialArrayGPU<<<blocks,threads>>>(d_ave_Byc_b, nz_, nx_, 1.0/1000.0);

    CHECK(hipMemcpy(d_Cp, h_Cp, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Cs, h_Cs, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_Den, h_Den, nz_ * nx_ * sizeof(float), hipMemcpyHostToDevice));

    moduliInit<<<blocks,threads>>>(d_Cp, d_Cs, d_Den, d_Lambda, d_Mu, nz_, nx_);
    aveMuInit<<<blocks,threads>>>(d_Mu, d_ave_Mu, nz_, nx_);
    aveBycInit<<<blocks,threads>>>(d_Den, d_ave_Byc_a, d_ave_Byc_b, nz_, nx_);

    CHECK(hipMemcpy(h_Lambda, d_Lambda, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_Mu, d_Mu, nz_ * nx_ * sizeof(float), hipMemcpyDeviceToHost));

}


Model::~Model() {
    free(h_Cp);
    free(h_Cs);
    free(h_Den);
    free(h_Lambda);
    free(h_Mu);
    free(h_CpGrad);
    CHECK(hipFree(d_Cp));
    CHECK(hipFree(d_Cs));
    CHECK(hipFree(d_Den));
    CHECK(hipFree(d_Lambda));
    CHECK(hipFree(d_Mu));
    CHECK(hipFree(d_ave_Mu));
    CHECK(hipFree(d_ave_Byc_a));
    CHECK(hipFree(d_ave_Byc_b));
    CHECK(hipFree(d_CpGrad));
}